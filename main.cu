
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

// Define the new polynomial function here
__device__ double polynomial(double x) {
    return x;
}

// CUDA kernel to compute the integral of a polynomial using the trapezoidal rule
__global__ void integratePolynomial(double a, double b, int num_intervals, double *result) {
    double h = (b - a) / num_intervals;
    double sum = 0.5 * (polynomial(a) + polynomial(b));

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = idx + 1; i < num_intervals; i += stride) {
        double x = a + i * h;
        sum += polynomial(x);
    }

    atomicAdd(reinterpret_cast<unsigned long long int*>(result), __double_as_longlong(sum * h));
}

int main() {
    // Define integration parameters
    double a = 0, b = 100; // Limits of integration
    int num_intervals = 1000000000; // Number of intervals for trapezoidal rule
    int answer = 50;

    double *d_result;
    hipMalloc(&d_result, sizeof(double));
    hipMemcpy(d_result, &answer, sizeof(double), hipMemcpyHostToDevice);

    // Launch kernel
    int blockSize = 256;
    int numBlocks = (num_intervals + blockSize - 1) / blockSize;

    integratePolynomial<<<numBlocks, blockSize>>>(a, b, num_intervals, d_result);

    double integral;
    hipMemcpy(&integral, d_result, sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_result);

    if (integral != answer) {
        std::cout << "Error: incorrect result: found " << integral << " instead of " << answer << std::endl;
    }

    // Output the result
    std::cout << "The integral of the polynomial in the range [" << a << ", " << b << "] is: " << integral << std::endl;

    return 0;
}
